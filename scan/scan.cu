#include "hip/hip_runtime.h"
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <driver_functions.h>

#include <thrust/scan.h>
#include <thrust/device_ptr.h>
#include <thrust/device_malloc.h>
#include <thrust/device_free.h>

#include "CycleTimer.h"

#define THREADS_PER_BLOCK 256


// helper function to round an integer up to the next power of 2
static inline int nextPow2(int n) {
    n--;
    n |= n >> 1;
    n |= n >> 2;
    n |= n >> 4;
    n |= n >> 8;
    n |= n >> 16;
    n++;
    return n;
}

__global__ void
scan_upsweep_kernel(int N, int two_d, int two_dplus1,int* result) {
    // compute overall thread index from position of thread in current
    // block, and given the block we are in (in this example only a 1D
    // calculation is needed so the code only looks at the .x terms of
    // blockDim and threadIdx.
    int index = blockIdx.x * blockDim.x + threadIdx.x;


    if(index < N && index % two_dplus1 == 0) {
      int index_twod_plus1 = index + two_dplus1 - 1;
      int index_twod_p = index  +two_d -1;
      result[index_twod_plus1] += result[index_twod_p];
    }
    // this check is necessary to make the code work for values of N
    // that are not a multiple of the thread block size (blockDim.x)
    //if (index < N)
    //   result[index] = alpha * x[index] + y[index];

}

__global__ void
scan_downsweep_kernel(int N, int two_d, int two_dplus1,int* result) {

    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < N && index % two_dplus1 == 0) {
      int index_twod = index + two_d - 1;
      int index_twod_plus = index + two_dplus1 -1;
      int t = result[index_twod];
      result[index_twod] = result[index_twod_plus];
      result[index_twod_plus] += t;
    }


}


// exclusive_scan --
//
// Implementation of an exclusive scan on global memory array `input`,
// with results placed in global memory `result`.
//
// N is the logical size of the input and output arrays, however
// students can assume that both the start and result arrays we
// allocated with next power-of-two sizes as described by the comments
// in cudaScan().  This is helpful, since your parallel scan
// will likely write to memory locations beyond N, but of course not
// greater than N rounded up to the next power of 2.
//
// Also, as per the comments in cudaScan(), you can implement an
// "in-place" scan, since the timing harness makes a copy of input and
// places it in result
void exclusive_scan(int* input, int N, int* result)
{

    // CS149 TODO:
    //
    // Implement your exclusive scan implementation here.  Keep in
    // mind that although the arguments to this function are device
    // allocated arrays, this is a function that is running in a thread
    // on the CPU.  Your implementation will need to make multiple calls
    // to CUDA kernel functions (that you must write) to implement the
    // scan.
  const int threadsPerBlock = 128;
  int new_N = nextPow2(N);
  hipMemset(result+N, 0, (new_N-N)*sizeof(int));
  

  const int blocks = (new_N + threadsPerBlock -1)/ threadsPerBlock;
  // int *arr = (int*)malloc(N * sizeof(int));
  // hipMemcpy(arr, input, N * sizeof(int), hipMemcpyDeviceToHost);
  // printf("input arr\n");
  // for(int i=0; i < N ;i ++) {
  //   printf("%d:%d ", i, arr[i]);
  // }
  // printf("\n");


  for(int two_d=1; two_d < new_N /2; two_d*=2) {
    int two_dplus1 = 2 * two_d;

      //double startTime = CycleTimer::currentSeconds();
  scan_upsweep_kernel<<<blocks, threadsPerBlock>>>(new_N, two_d, two_dplus1, result);
    hipDeviceSynchronize();
      //double endTime = CycleTimer::currentSeconds();

  }
  // hipMemcpy(arr, result, N*sizeof(int), hipMemcpyDeviceToHost);
  // printf("upsweep cuda\n");
  // for(int i=0; i < N ;i ++) {
  //   printf("%d:%d ", i, arr[i]);
  // }
  // printf("\n");


    hipMemset(result + new_N-1, 0, sizeof(int));

  for(int two_d=new_N/2; two_d >= 1; two_d/=2) {
    int two_dplus1 = 2 * two_d;
  scan_downsweep_kernel<<<blocks, threadsPerBlock>>>(new_N, two_d, two_dplus1, result);
    hipDeviceSynchronize();
  }
  // hipDeviceSynchronize();
  // hipMemcpy(arr, result, N * sizeof(int), hipMemcpyDeviceToHost);
  // printf("downsweep cuda\n");
  // for(int i=0; i < N ;i ++) {
  //   printf("%d:%d ", i, arr[i]);
  // }
  // printf("\n");
  // free(arr);

}


//
// cudaScan --
//
// This function is a timing wrapper around the student's
// implementation of scan - it copies the input to the GPU
// and times the invocation of the exclusive_scan() function
// above. Students should not modify it.
double cudaScan(int* inarray, int* end, int* resultarray)
{
    int* device_result;
    int* device_input;
    int N = end - inarray;  

    // This code rounds the arrays provided to exclusive_scan up
    // to a power of 2, but elements after the end of the original
    // input are left uninitialized and not checked for correctness.
    //
    // Student implementations of exclusive_scan may assume an array's
    // allocated length is a power of 2 for simplicity. This will
    // result in extra work on non-power-of-2 inputs, but it's worth
    // the simplicity of a power of two only solution.

    int rounded_length = nextPow2(end - inarray);
  // printf("input array\n");
  // for(int i=0; i < (end- inarray ); i++ ) {
  //   printf("%d:%d ", i, inarray[i]);
  // }
  // printf("\n");
    
    hipMalloc((void **)&device_result, sizeof(int) * rounded_length);
    hipMalloc((void **)&device_input, sizeof(int) * rounded_length);

    // For convenience, both the input and output vectors on the
    // device are initialized to the input values. This means that
    // students are free to implement an in-place scan on the result
    // vector if desired.  If you do this, you will need to keep this
    // in mind when calling exclusive_scan from find_repeats.
    hipMemcpy(device_input, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(device_result, inarray, (end - inarray) * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    exclusive_scan(device_input, N, device_result);

    // Wait for completion
    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
       
    hipMemcpy(resultarray, device_result, (end - inarray) * sizeof(int), hipMemcpyDeviceToHost);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


// cudaScanThrust --
//
// Wrapper around the Thrust library's exclusive scan function
// As above in cudaScan(), this function copies the input to the GPU
// and times only the execution of the scan itself.
//
// Students are not expected to produce implementations that achieve
// performance that is competition to the Thrust version, but it is fun to try.
double cudaScanThrust(int* inarray, int* end, int* resultarray) {

    int length = end - inarray;
    thrust::device_ptr<int> d_input = thrust::device_malloc<int>(length);
    thrust::device_ptr<int> d_output = thrust::device_malloc<int>(length);
    
    hipMemcpy(d_input.get(), inarray, length * sizeof(int), hipMemcpyHostToDevice);

    double startTime = CycleTimer::currentSeconds();

    thrust::exclusive_scan(d_input, d_input + length, d_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();
   
    hipMemcpy(resultarray, d_output.get(), length * sizeof(int), hipMemcpyDeviceToHost);

    thrust::device_free(d_input);
    thrust::device_free(d_output);

    double overallDuration = endTime - startTime;
    return overallDuration; 
}


__global__ void 
flag_repeats_kernel(int* input, int* output, int N) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < N-1 && input[index] == input[index+1]) {
    output[index] = 1;
  }


}

__global__ void 
flags_extract_indices(int *input, int* output, int N ) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < N-1 && input[index] < input[index+1]) {
    output[input[index]] = index;
  }
}

// find_repeats --
//
// Given an array of integers `device_input`, returns an array of all
// indices `i` for which `device_input[i] == device_input[i+1]`.
//
// Returns the total number of pairs found
int find_repeats(int* device_input, int length, int* device_output) {

    // CS149 TODO:
    //
    // Implement this function. You will probably want to
    // make use of one or more calls to exclusive_scan(), as well as
    // additional CUDA kernel launches.
    //    
    // Note: As in the scan code, the calling code ensures that
    // allocated arrays are a power of 2 in size, so you can use your
    // exclusive_scan function with them. However, your implementation
    // must ensure that the results of find_repeats are correct given
    // the actual array length.
  int *flags_arr;
  int *flags_sum_arr;

  hipMalloc((void**)&flags_arr, length * sizeof(int));
  hipMalloc((void**)&flags_sum_arr, length * sizeof(int));

  const int threadsPerBlock = 512;
  const int blocks = (length + threadsPerBlock -1 ) / threadsPerBlock;
  int repeat_indices_count;

  flag_repeats_kernel<<<blocks, threadsPerBlock>>>(device_input, flags_arr, length);
  exclusive_scan(flags_arr, length, flags_sum_arr);
  flags_extract_indices<<<blocks, threadsPerBlock>>>(flags_sum_arr, device_output, length);
  hipMemcpy(&repeat_indices_count, flags_sum_arr+length-1, 1, hipMemcpyDeviceToHost);

  return repeat_indices_count; 

}


//
// cudaFindRepeats --
//
// Timing wrapper around find_repeats. You should not modify this function.
double cudaFindRepeats(int *input, int length, int *output, int *output_length) {

    int *device_input;
    int *device_output;
    int rounded_length = nextPow2(length);
    
    hipMalloc((void **)&device_input, rounded_length * sizeof(int));
    hipMalloc((void **)&device_output, rounded_length * sizeof(int));
    hipMemcpy(device_input, input, length * sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    double startTime = CycleTimer::currentSeconds();
    
    int result = find_repeats(device_input, length, device_output);

    hipDeviceSynchronize();
    double endTime = CycleTimer::currentSeconds();

    // set output count and results array
    *output_length = result;
    hipMemcpy(output, device_output, length * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(device_input);
    hipFree(device_output);

    float duration = endTime - startTime; 
    return duration;
}



void printCudaInfo()
{
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    printf("---------------------------------------------------------\n");
    printf("Found %d CUDA devices\n", deviceCount);

    for (int i=0; i<deviceCount; i++)
    {
        hipDeviceProp_t deviceProps;
        hipGetDeviceProperties(&deviceProps, i);
        printf("Device %d: %s\n", i, deviceProps.name);
        printf("   SMs:        %d\n", deviceProps.multiProcessorCount);
        printf("   Global mem: %.0f MB\n",
               static_cast<float>(deviceProps.totalGlobalMem) / (1024 * 1024));
        printf("   CUDA Cap:   %d.%d\n", deviceProps.major, deviceProps.minor);
    }
    printf("---------------------------------------------------------\n"); 
}
